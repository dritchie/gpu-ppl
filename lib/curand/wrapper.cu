
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"


extern "C"
{
	__device__ void rand_init(unsigned long long seed,
							  unsigned long long subsequence,
							  unsigned long long offset,
							  hiprandStateXORWOW* state)
	{
		hiprand_init(seed, subsequence, offset, state);
	}

	__device__ double rand_uniform(hiprandStateXORWOW* state)
	{
		return hiprand_uniform_double(state);
	}
}