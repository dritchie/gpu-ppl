
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"


extern "C"
{
	__device__ void cu_rand_init(unsigned long long seed,
							  unsigned long long subsequence,
							  unsigned long long offset,
							  hiprandStateXORWOW* state)
	{
		hiprand_init(seed, subsequence, offset, state);
	}

	__device__ double cu_rand_uniform(hiprandStateXORWOW* state)
	{
		return hiprand_uniform_double(state);
	}
}