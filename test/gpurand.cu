
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

#define seed 42

__global__ void kernel(double* outdata)
{
	// curandStateXORWOW_t state;
	hiprandStateMRG32k3a_t state;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state);
	outdata[idx] = hiprand_uniform(&state);
}

int main()
{
	double* data;
	hipMalloc((void**)data, sizeof(double));
	kernel<<<1,1>>>(data);
	hipFree(data);
	return 0;
}